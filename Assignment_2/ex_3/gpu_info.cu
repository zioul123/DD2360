/*
Running this flle will print the GOU info. Code taken from:
https://devblogs.nvidia.com/how-query-device-properties-and-handle-errors-cuda-cc/
*/
#include <stdio.h>

int main() {
    int nDevices;

    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n",
        prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
        prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
        2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    }
}
